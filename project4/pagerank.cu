#include "hip/hip_runtime.h"
__device__ __constant__ int c_num_nodes;
__global__
void device_graph_propagate(const uint* graph_indices, 
        const uint* graph_edges, 
        const float* graph_nodes_in, 
        float* graph_nodes_out, 
        const float* inv_edges_per_node)
        //,int num_nodes) 
{

    // TODO: fill in the kernel code here
    int node_index = blockIdx.x*blockDim.x + threadIdx.x;
    if(node_index < c_num_nodes){
        float sum = 0.f;
        for(uint j = graph_indices[node_index]; j < graph_indices[node_index+1]; j++) {
            sum += graph_nodes_in[graph_edges[j]]*inv_edges_per_node[graph_edges[j]];
        }
        graph_nodes_out[node_index] = 0.5f/(float)c_num_nodes+0.5f*sum;
    }
}

double device_graph_iterate(const uint* h_graph_indices
        , const uint* h_graph_edges
        , const float* h_node_values_input
        , float* h_gpu_node_values_output
        , const float* h_inv_edges_per_node
        , int nr_iterations
        , int num_nodes
        , int avg_edges) {
    // TODO: allocate GPU memory
    int err_1 = hipSuccess;
    int err_2 = hipSuccess;
    int err_3 = hipSuccess;
    int err_4 = hipSuccess;
    int err_5 = hipSuccess;
    //int err_6 = hipSuccess;

    float *cuda_1;
    float *cuda_2;
    uint *cuda_graph_indices;
    uint *cuda_graph_edges;
    float *cuda_inv_edges_per_node;
    //int *cuda_num_nodes;

    err_1 = hipMalloc((void **)&cuda_1, num_nodes*sizeof(float));
    err_2 = hipMalloc((void **)&cuda_2, num_nodes*sizeof(float));
    err_3 = hipMalloc((void **)&cuda_graph_indices, (num_nodes+1)*sizeof(uint));
    err_4 = hipMalloc((void **)&cuda_graph_edges, (num_nodes*avg_edges)*sizeof(uint));
    err_5 = hipMalloc((void **)&cuda_inv_edges_per_node, (num_nodes)*sizeof(float));
    //err_6 = hipMalloc((void **)&cuda_num_nodes, sizeof(int));
    // TODO: check for allocation failure
    if(err_1) throw err_1;
    if(err_2) throw err_2;
    if(err_3) throw err_3;
    if(err_4) throw err_4;
    if(err_5) throw err_5;
    //if(err_6) throw err_6;

    // TODO: copy data to the GPU
    hipMemcpy(cuda_1, h_node_values_input, num_nodes*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cuda_2, h_node_values_input, num_nodes*sizeof(float), hipMemcpyHostToDevice); 
    hipMemcpy(cuda_graph_indices, h_graph_indices, (num_nodes+1)*sizeof(uint), hipMemcpyHostToDevice);
    hipMemcpy(cuda_graph_edges, h_graph_edges, (num_nodes*avg_edges)*sizeof(uint), hipMemcpyHostToDevice);
    hipMemcpy(cuda_inv_edges_per_node, h_inv_edges_per_node, (num_nodes)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_num_nodes), &num_nodes, sizeof(int), 0, hipMemcpyHostToDevice);
    //hipMemcpy(cuda_num_nodes, &num_nodes, sizeof(int), hipMemcpyHostToDevice);
    start_timer(&timer);

    const int thread_num = 1024; // Should be divisor of 32768.
    //GTX 1060, Maximum thread_num is 1024.
    int block_size = 512;//(num_nodes)/thread_num;

    // TODO: launch your kernels the appropriate number of iterations
    for(int iter = 0; iter < nr_iterations/2 ; iter++) {
            device_graph_propagate<<<block_size, thread_num>>>(cuda_graph_indices, cuda_graph_edges, cuda_1, cuda_2, cuda_inv_edges_per_node);//, num_nodes);
            device_graph_propagate<<<block_size, thread_num>>>(cuda_graph_indices, cuda_graph_edges, cuda_2, cuda_1, cuda_inv_edges_per_node);//, num_nodes);
    }

    check_launch("gpu graph propagate");
    double gpu_elapsed_time = stop_timer(&timer);
    // TODO: copy final data back to the host for correctness checking
    if(nr_iterations % 2) {
        device_graph_propagate<<<block_size, thread_num>>>(cuda_graph_indices, cuda_graph_edges, cuda_1, cuda_2, cuda_inv_edges_per_node);//, num_nodes);
        hipMemcpy(h_gpu_node_values_output, cuda_2, num_nodes*sizeof(float), hipMemcpyDeviceToHost);
    } else {
        hipMemcpy(h_gpu_node_values_output, cuda_1, num_nodes*sizeof(float), hipMemcpyDeviceToHost);
    }
    // TODO: free the memory you allocated!
    hipDeviceSynchronize();
    hipFree(cuda_1);
    hipFree(cuda_2);
    hipFree(cuda_graph_indices);
    hipFree(cuda_graph_edges);
    hipFree(cuda_inv_edges_per_node);

    return gpu_elapsed_time;
}
